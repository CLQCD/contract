#include <baryon.cuh>

void baryon_in_jk_ml(void *correl, void *propag_i, void *propag_j, void *propag_m, size_t volume, int gamma_ij,
                     int gamma_kl, int gamma_mn)
{
  unsigned int grid = (volume * (Ns * Ns) + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int block = BLOCK_SIZE;
  dim3 gridDim(grid, 1, 1);
  dim3 blockDim(block, 1, 1);

  Arguments args_h = {correl, propag_j, propag_m, propag_i, volume, gamma_ij, gamma_kl, gamma_mn};
  CUDA_ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(args), &args_h, sizeof(Arguments)));
  CUDA_ERROR_CHECK(hipLaunchKernel(instantiate<IN_JK_ML>(gamma_kl), gridDim, blockDim, {}));

  return;
}
