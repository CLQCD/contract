#include <baryon_v2.cuh>
#include <baryon.h>

void baryon_il_jk_mn(void *correl, void *propag_i, void *propag_j, void *propag_m, size_t volume, int gamma_ij,
                     int gamma_kl, int gamma_mn)
{
  if (volume % TILE_SIZE != 0) {
    fprintf(stderr, "Error: Volume must be a multiple of TILE_SIZE\n");
    exit(-1);
  }

  unsigned int grid = (volume * (Ns * Ns) + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int block = BLOCK_SIZE;
  dim3 gridDim(grid, 1, 1);
  dim3 blockDim(block, 1, 1);

  Arguments args_h = {correl, propag_j, propag_i, propag_m, volume, gamma_ij, gamma_kl, gamma_mn};
  CUDA_ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(args), &args_h, sizeof(Arguments)));
  CUDA_ERROR_CHECK(hipLaunchKernel(instantiate<IL_JK_MN>(gamma_mn), gridDim, blockDim, {}));

  return;
}
