#include <baryon.cuh>

void baryon_ik_jn_ml(void *correl, void *propag_i, void *propag_j, void *propag_m, size_t volume, int gamma_ij,
                     int gamma_kl, int gamma_mn)
{
  unsigned int grid = (volume * (Ns * Ns) + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int block = BLOCK_SIZE;
  dim3 gridDim(grid, 1, 1);
  dim3 blockDim(block, 1, 1);

  Arguments args_h = {correl, propag_i, propag_m, propag_j, volume, gamma_ij, gamma_kl, gamma_mn};
  CUDA_ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(args), &args_h, sizeof(Arguments)));

  CUDA_ERROR_CHECK(hipLaunchKernel(instantiate<IK_JN_ML>(gamma_kl), gridDim, blockDim, {}));
}
