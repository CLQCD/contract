#include <hip/hip_runtime.h>
#include <contract.h>
#include <baryon.h>
#include <meson.h>

void meson_two_point(void *correl, void *propag_a, void *propag_b, size_t volume, int gamma_ab, int gamma_dc)
{
  meson::launch(correl, propag_a, propag_b, volume, gamma_ab, gamma_dc);
  return;
}

void meson_all_source_two_point(void **correl, void *propag_a, void *propag_b, size_t volume, int gamma_ab)
{
  meson_all_source::launch(correl, propag_a, propag_b, volume, gamma_ab);
  return;
}

void meson_all_sink_two_point(void **correl, void *propag_a, void *propag_b, size_t volume, int gamma_dc)
{
  meson_all_sink::launch(correl, propag_a, propag_b, volume, gamma_dc);
  return;
}

void baryon_two_point(void *correl, void *propag_a, void *propag_b, void *propag_c, BaryonContractType contract_type,
                      size_t volume, int gamma_ab, int gamma_de, int gamma_fc)
{
  switch (contract_type) {
  case AD_BE_CF:
    baryon::launch<AD_BE_CF>(correl, propag_a, propag_b, propag_c, volume, gamma_ab, gamma_de, gamma_fc);
    break;
  case AD_BF_CE:
    baryon::launch<AD_BF_CE>(correl, propag_a, propag_b, propag_c, volume, gamma_ab, gamma_de, gamma_fc);
    break;
  case AE_BD_CF:
    baryon::launch<AE_BD_CF>(correl, propag_a, propag_b, propag_c, volume, gamma_ab, gamma_de, gamma_fc);
    break;
  case AE_BF_CD:
    baryon::launch<AE_BF_CD>(correl, propag_a, propag_b, propag_c, volume, gamma_ab, gamma_de, gamma_fc);
    break;
  case AF_BD_CE:
    baryon::launch<AF_BD_CE>(correl, propag_a, propag_b, propag_c, volume, gamma_ab, gamma_de, gamma_fc);
    break;
  case AF_BE_CD:
    baryon::launch<AF_BE_CD>(correl, propag_a, propag_b, propag_c, volume, gamma_ab, gamma_de, gamma_fc);
    break;
  default: break;
  }
  return;
}

void baryon_two_point_v2(void *correl, void *propag_i, void *propag_j, void *propag_m, BaryonContractType contract_type,
                         size_t volume, int gamma_ij, int gamma_kl, int gamma_mn)
{
  switch (contract_type) {
  case IK_JL_MN: baryon_ik_jl_mn(correl, propag_i, propag_j, propag_m, volume, gamma_ij, gamma_kl, gamma_mn); break;
  case IK_JN_ML: baryon_ik_jn_ml(correl, propag_i, propag_j, propag_m, volume, gamma_ij, gamma_kl, gamma_mn); break;
  case IL_JK_MN: baryon_il_jk_mn(correl, propag_i, propag_j, propag_m, volume, gamma_ij, gamma_kl, gamma_mn); break;
  case IL_JN_MK: baryon_il_jn_mk(correl, propag_i, propag_j, propag_m, volume, gamma_ij, gamma_kl, gamma_mn); break;
  case IN_JK_ML: baryon_in_jk_ml(correl, propag_i, propag_j, propag_m, volume, gamma_ij, gamma_kl, gamma_mn); break;
  case IN_JL_MK: baryon_in_jl_mk(correl, propag_i, propag_j, propag_m, volume, gamma_ij, gamma_kl, gamma_mn); break;
  default: break;
  }
  return;
}

void init(int device)
{
  CUDA_ERROR_CHECK(hipSetDevice(device));
  return;
}